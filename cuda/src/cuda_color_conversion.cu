#include "hip/hip_runtime.h"
#include "cuda_color_conversion.cuh"
#include <hip/hip_runtime.h>
#include <>

namespace evmcuda {

// RGB to YIQ conversion matrix stored in constant memory for fast access
__constant__ float c_rgb2yiq_matrix[9] = {
    0.299f,       0.587f,       0.114f,         // Y coefficients
    0.59590059f, -0.27455667f, -0.32134392f,   // I coefficients  
    0.21153661f, -0.52273617f,  0.31119955f    // Q coefficients
};

// YIQ to RGB conversion matrix stored in constant memory for fast access
__constant__ float c_yiq2rgb_matrix[9] = {
    1.0f,        0.9559863f,   0.6208248f,     // R coefficients
    1.0f,       -0.2720128f,  -0.6472042f,    // G coefficients
    1.0f,       -1.1067402f,   1.7042304f     // B coefficients
};

__global__ void rgb_to_yiq_kernel(const float3* rgb_data, float3* yiq_data, int num_pixels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < num_pixels) {
        float3 rgb = rgb_data[idx];
        float3 yiq;
        
        // Matrix multiplication: YIQ = RGB2YIQ_MATRIX * RGB
        yiq.x = c_rgb2yiq_matrix[0] * rgb.x + c_rgb2yiq_matrix[1] * rgb.y + c_rgb2yiq_matrix[2] * rgb.z; // Y
        yiq.y = c_rgb2yiq_matrix[3] * rgb.x + c_rgb2yiq_matrix[4] * rgb.y + c_rgb2yiq_matrix[5] * rgb.z; // I
        yiq.z = c_rgb2yiq_matrix[6] * rgb.x + c_rgb2yiq_matrix[7] * rgb.y + c_rgb2yiq_matrix[8] * rgb.z; // Q
        
        yiq_data[idx] = yiq;
    }
}

__global__ void yiq_to_rgb_kernel(const float3* yiq_data, float3* rgb_data, int num_pixels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < num_pixels) {
        float3 yiq = yiq_data[idx];
        float3 rgb;
        
        // Matrix multiplication: RGB = YIQ2RGB_MATRIX * YIQ
        rgb.x = c_yiq2rgb_matrix[0] * yiq.x + c_yiq2rgb_matrix[1] * yiq.y + c_yiq2rgb_matrix[2] * yiq.z; // R
        rgb.y = c_yiq2rgb_matrix[3] * yiq.x + c_yiq2rgb_matrix[4] * yiq.y + c_yiq2rgb_matrix[5] * yiq.z; // G
        rgb.z = c_yiq2rgb_matrix[6] * yiq.x + c_yiq2rgb_matrix[7] * yiq.y + c_yiq2rgb_matrix[8] * yiq.z; // B
        
        rgb_data[idx] = rgb;
    }
}

hipError_t rgb_to_yiq(const float3* rgb_data, float3* yiq_data, int width, int height) {
    int num_pixels = width * height;
    
    // Define block size and grid size
    int block_size = 256;
    int grid_size = (num_pixels + block_size - 1) / block_size;
    
    // Launch kernel
    rgb_to_yiq_kernel<<<grid_size, block_size>>>(rgb_data, yiq_data, num_pixels);
    
    // Check for kernel launch errors
    return hipGetLastError();
}

hipError_t yiq_to_rgb(const float3* yiq_data, float3* rgb_data, int width, int height) {
    int num_pixels = width * height;
    
    // Define block size and grid size
    int block_size = 256;
    int grid_size = (num_pixels + block_size - 1) / block_size;
    
    // Launch kernel
    yiq_to_rgb_kernel<<<grid_size, block_size>>>(yiq_data, rgb_data, num_pixels);
    
    // Check for kernel launch errors
    return hipGetLastError();
}

} // namespace evmcuda